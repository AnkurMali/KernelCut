#include "hip/hip_runtime.h"
#include "gpufiltering.h"
#define SAMPLE_STEP 8
// k_p(p,q)
__global__ void filtering_kernel_p(float * X, float * values, float * sigmas, float * output, size_t N) 
{
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    float p_c[3];
    float sigmasquared;
    if(idx>=N)
        return;
    p_c[0] = X[idx*3];
    p_c[1] = X[idx*3+1];
    p_c[2] = X[idx*3+2];
    sigmasquared = sigmas[idx]*sigmas[idx]*(-2.0);
    output[idx] = 1;
    float diff = 0;
    float out = 0;
    for(int n=0;n<N;n+=SAMPLE_STEP){
        diff=(p_c[0]-X[n*3])*(p_c[0]-X[n*3])+(p_c[1]-X[n*3+1])*(p_c[1]-X[n*3+1])+(p_c[2]-X[n*3+2])*(p_c[2]-X[n*3+2]);
        out += __expf(diff/sigmasquared)*values[n];
    }
    output[idx] = out*SAMPLE_STEP;
}

// k_q(p,q)
__global__ void filtering_kernel_q(float * X, float * values, float * sigmas, float * output, size_t N) 
{
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    float p_c[3];
    if(idx>=N)
        return;
    p_c[0] = X[idx*3];
    p_c[1] = X[idx*3+1];
    p_c[2] = X[idx*3+2];
    output[idx] = 1;
    float diff = 0;
    float out = 0;
    for(int n=0;n<N;n+=SAMPLE_STEP){
        diff=(p_c[0]-X[n*3])*(p_c[0]-X[n*3])+(p_c[1]-X[n*3+1])*(p_c[1]-X[n*3+1])+(p_c[2]-X[n*3+2])*(p_c[2]-X[n*3+2]);
        out += __expf(diff/sigmas[n]/sigmas[n]/(-2.0))*values[n];
    }
    output[idx] = out*SAMPLE_STEP;
}

// k_pq(p,q)
__global__ void filtering_kernel_pq(float * X, float * values, float * sigmas, float * output, size_t N) 
{
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    float p_c[3];
    if(idx>=N)
        return;
    p_c[0] = X[idx*3];
    p_c[1] = X[idx*3+1];
    p_c[2] = X[idx*3+2];
    output[idx] = 1;
    float diff = 0;
    float out = 0;
    float sigma_p = sigmas[idx];
    for(int n=0;n<N;n+=SAMPLE_STEP){
        diff=(p_c[0]-X[n*3])*(p_c[0]-X[n*3])+(p_c[1]-X[n*3+1])*(p_c[1]-X[n*3+1])+(p_c[2]-X[n*3+2])*(p_c[2]-X[n*3+2]);
        out += __expf(diff/sigmas[n]/sigma_p/(-2.0))*values[n];
    }
    output[idx] = out*SAMPLE_STEP;
}

int gpufilteringdemo(int N) {
    cout << "filtering for N = " <<N<< endl; // prints !!!Hello World!!!
    srand(time(NULL));
    clock_t startTime,endTime;
    // intialization
    startTime = clock();
    float * X = new float[N*3];
    float * values = new float[N];
    float * sigmas = new float[N];
    for(int n=0;n<N;n++){
        X[n*3] = (rand()%1000)/1000.0;
        X[n*3+1] = (rand()%1000)/1000.0;
        X[n*3+2] = (rand()%1000)/1000.0;
        values[n] = (rand()%1000)/1000.0;
        sigmas[n] = 0.01+(rand()%100)/100000.0;
    }
    endTime = clock();
    cout << "Initialization: "<<float( endTime- startTime ) / (float)CLOCKS_PER_SEC<< " seconds." << endl;
    // parallel dense filtering
    startTime = clock();
    float *Xd, *valuesd, *sigmasd, * outputd;
    hipMalloc((void **)&Xd, sizeof(float)*N*3);
    hipMalloc((void **)&valuesd, sizeof(float)*N);
    hipMalloc((void **)&sigmasd, sizeof(float)*N);
    hipMalloc((void **)&outputd, sizeof(float)*N);
    hipMemcpy(Xd, X, sizeof(float)*N*3, hipMemcpyHostToDevice);
    hipMemcpy(valuesd, values, sizeof(float)*N, hipMemcpyHostToDevice);
    hipMemcpy(sigmasd, sigmas, sizeof(float)*N, hipMemcpyHostToDevice);
    endTime = clock();
    cout << "memory copy: "<<float( endTime- startTime ) / (float)CLOCKS_PER_SEC<< " seconds." << endl;
    int num_block = ((N%BLOCK_SIZE)==0)?(N/BLOCK_SIZE):(N/BLOCK_SIZE+1);
    startTime = clock();
    cout<<"number of blocks: "<<num_block<<endl;
    // launch the kernel
    filtering_kernel_q<<<num_block, BLOCK_SIZE>>>(Xd, valuesd, sigmasd, outputd, N);
    hipDeviceSynchronize();
    hipError_t code = hipGetLastError();
    if(code!=hipSuccess){
        printf("Cuda error -- %s\n",hipGetErrorString(code));
    } else printf("Cuda success\n");
    float * para_output = new float[N];
    hipMemcpy(para_output, outputd, sizeof(float)*N, hipMemcpyDeviceToHost);
    endTime = clock();
    float kernel1_time = float( endTime- startTime ) / (float)CLOCKS_PER_SEC;
    cout << "kernel time : "<<kernel1_time<< " seconds." << endl;

    delete [] X;
    delete [] values;
    delete [] sigmas;
    delete [] para_output;
    hipFree(Xd);
    hipFree(valuesd);
    hipFree(sigmasd);
    hipFree(outputd);
    printf("End of program.\n");
    return 0;
}

float * gpufiltering(float * X, float * values, float * sigmas, size_t N, char flag)
{
	//int devicecount=0;
	//hipGetDeviceCount(&devicecount);
	//printf("devicec count %d\n",devicecount);
    float * output = new float[N];
    float *Xd, *valuesd, *sigmasd, * outputd;
    hipMalloc((void **)&Xd, sizeof(float)*N*3);
    hipMalloc((void **)&valuesd, sizeof(float)*N);
    hipMalloc((void **)&sigmasd, sizeof(float)*N);
    hipMalloc((void **)&outputd, sizeof(float)*N);
    hipMemcpy(Xd, X, sizeof(float)*N*3, hipMemcpyHostToDevice);
    hipMemcpy(valuesd, values, sizeof(float)*N, hipMemcpyHostToDevice);
    hipMemcpy(sigmasd, sigmas, sizeof(float)*N, hipMemcpyHostToDevice);
    int num_block = ((N%BLOCK_SIZE)==0)?(N/BLOCK_SIZE):(N/BLOCK_SIZE+1);
    // launch the kernel
    if(flag=='p')
        filtering_kernel_p<<<num_block, BLOCK_SIZE>>>(Xd, valuesd, sigmasd, outputd, N);
    else if(flag=='q')
        filtering_kernel_q<<<num_block, BLOCK_SIZE>>>(Xd, valuesd, sigmasd, outputd, N);
    else if(flag=='m')
        filtering_kernel_pq<<<num_block, BLOCK_SIZE>>>(Xd, valuesd, sigmasd, outputd, N);
    hipDeviceSynchronize();
    hipError_t code = hipGetLastError();
    if(code!=hipSuccess){
        printf("Cuda error -- %s\n",hipGetErrorString(code));
    } //else printf("Cuda success\n");
    hipMemcpy(output, outputd, sizeof(float)*N, hipMemcpyDeviceToHost);
    // deallocation
    hipFree(Xd);
    hipFree(valuesd);
    hipFree(sigmasd);
    hipFree(outputd);
    return output;
}
